#include "hip/hip_runtime.h"
/**
 * Name: Seoyoung Park
 * Student id:
 * ITSC email: sparkap@connect.ust.hk
 */

#include <cstring>
#include <cstdint>
#include <cstdlib>

#include <vector>
#include <iostream>

#include "cuda_push_relabel.h"

using namespace std;

void pre_flow(int *dist, int64_t *excess, int *cap, int *flow, int N, int src) {
    dist[src] = N;

    for (auto v = 0; v < N; v++) {
        flow[utils::idx(src, v, N)] = cap[utils::idx(src, v, N)];
        flow[utils::idx(v, src, N)] = -flow[utils::idx(src, v, N)];
        excess[v] = flow[utils::idx(src, v, N)];
    }
}

// Device code
__device__ int idx(int x, int y, int n) {
    return x * n + y;
}

__global__ void push(int *active_nodes, int active_node_size, int N, int *cap, int *flow, int *dist, int64_t *excess, int *stash_send){
    int blockId = blockIdx.x;
    int threadId = threadIdx.x;
    int numBlock = gridDim.x;
    int numThread = blockDim.x;
    int v, u;
    extern __shared__ unsigned long long residual_cap[];

    for (int i = blockId; i < active_node_size; i+=numBlock){
        u = active_nodes[i];

        for (v = threadId; v < N; v+=numThread){
            residual_cap[v] = cap[idx(u, v, N)] - flow[idx(u, v, N)];
        }
        //__syncthreads();

        if (threadId == 0){
            v = 0;

            while(excess[u]>0 && v < N){
                if (residual_cap[v] > 0 && dist[u] > dist[v]){
                    stash_send[idx(u, v, N)] = min((unsigned long long)excess[u], residual_cap[v]);
                    excess[u] -= stash_send[idx(u, v, N)];
                }
                v++;
            }
        }
        //__syncthreads();
    }
}


int push_relabel(int blocks_per_grid, int threads_per_block, int N, int src, int sink, int *cap, int *flow) {
    size_t size_in_int = N * sizeof(int);
    size_t size_n_int64_t = N * sizeof(int64_t);

    // Allocate Vectors in host memory (CPU)
    int *h_dist = (int *) calloc(N, sizeof(int));
    int *h_stash_dist = (int *) calloc(N, sizeof(int));
    auto *h_excess = (int64_t *) calloc(N, sizeof(int64_t));
    auto *h_stash_excess = (int64_t *) calloc(N, sizeof(int64_t));
    vector<int> h_active_nodes;
    int *h_stash_send = (int *) calloc(N * N, sizeof(int));

    //printf("%d %d\n", sizeof(int), sizeof(int64_t));

    // Allocate Vectors in device memory (GPU)
    int64_t *d_excess, *d_stash_excess;
    int *d_dist, *d_stash_dist, *d_stash_send, *d_cap, *d_flow;

    hipMalloc((void**) &d_dist, size_in_int);
    hipMalloc((void**) &d_excess, size_n_int64_t);
    hipMalloc((void**) &d_stash_dist, size_in_int);
    hipMalloc((void**) &d_stash_send, size_in_int);
    hipMalloc((void**) &d_stash_excess, size_n_int64_t);
    hipMalloc((void**) &d_cap, N * N * sizeof(int));
    hipMalloc((void**) &d_flow, N * N * sizeof(int));


    // Initialise input data
    // PreFlow.
    pre_flow(h_dist, h_excess, cap, flow, N, src);

    for (auto u = 0; u < N; u++) {
        if (u != src && u != sink) {
            h_active_nodes.emplace_back(u);
        }
    }

    int* d_active_nodes = &h_active_nodes[0];

    hipMalloc((void**) &d_active_nodes, h_active_nodes.size() * sizeof(int));

    // Copy vectors from host memory to device memory
    hipMemcpy(d_dist, h_dist, size_in_int, hipMemcpyHostToDevice);
    hipMemcpy(d_excess, h_excess, size_n_int64_t, hipMemcpyHostToDevice);
    hipMemcpy(d_stash_dist, h_stash_dist, size_in_int, hipMemcpyHostToDevice);
    hipMemcpy(d_stash_send, h_stash_send, size_in_int, hipMemcpyHostToDevice);
    hipMemcpy(d_stash_excess, h_stash_excess, size_n_int64_t, hipMemcpyHostToDevice);
    hipMemcpy(d_cap, cap, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_flow, flow, N * N *  sizeof(int), hipMemcpyHostToDevice);

    // Four-Stage Pulses.
    while (!h_active_nodes.empty()) {
        // Stage 1: push.
        //foo_kernel<<<blocks_per_grid, threads_per_block>>>(d_dist, d_excess);
        hipMemcpy(d_active_nodes, h_active_nodes.data(), sizeof(int) * h_active_nodes.size(), hipMemcpyHostToDevice);
        push<<<blocks_per_grid, threads_per_block>>>(d_active_nodes, h_active_nodes.size(), N, d_cap, d_flow, d_dist, d_excess, d_stash_send);
        /*
        for (auto u : h_active_nodes) {
            for (auto v = 0; v < N; v++) {
                auto residual_cap = cap[utils::idx(u, v, N)] - flow[utils::idx(u, v, N)];

                if (residual_cap > 0 && h_dist[u] > h_dist[v] && h_excess[u] > 0) {
                    h_stash_send[utils::idx(u, v, N)] = std::min<int64_t>(h_excess[u], residual_cap);
                    h_excess[u] -= h_stash_send[utils::idx(u, v, N)];
                }

                if (h_stash_send[utils::idx(u, v, N)] > 0) {
                    flow[utils::idx(u, v, N)] += h_stash_send[utils::idx(u, v, N)];
                    flow[utils::idx(v, u, N)] -= h_stash_send[utils::idx(u, v, N)];
                    h_stash_excess[v] += h_stash_send[utils::idx(u, v, N)];
                    h_stash_send[utils::idx(u, v, N)] = 0;
                }
            }
        }
        */

        // Copy result
        //hipMemcpy(h_active_nodes, d_active_nodes, size_in_int, hipMemcpyDeviceToHost);
        hipMemcpy(cap, d_cap, size_in_int, hipMemcpyDeviceToHost);
        hipMemcpy(flow, d_flow, size_in_int, hipMemcpyDeviceToHost);
        hipMemcpy(h_dist, d_dist, size_in_int, hipMemcpyDeviceToHost);
        hipMemcpy(h_excess, d_excess, size_in_int, hipMemcpyDeviceToHost);
        hipMemcpy(h_stash_send, d_stash_send, size_in_int, hipMemcpyDeviceToHost);

        // Stage 2: relabel (update dist to stash_dist).
        memcpy(h_stash_dist, h_dist, N * sizeof(int));
        for (auto u : h_active_nodes) {
            if (h_excess[u] > 0) {
                int min_dist = INT32_MAX;
                for (auto v = 0; v < N; v++) {
                    auto residual_cap = cap[utils::idx(u, v, N)] - flow[utils::idx(u, v, N)];
                    if (residual_cap > 0) {
                        min_dist = min(min_dist, h_dist[v]);
                        h_stash_dist[u] = min_dist + 1;
                    }
                }
            }
        }

        // Stage 3: update dist.
        swap(h_dist, h_stash_dist);

        // Stage 4: apply excess-flow changes for destination vertices.
        for (auto v = 0; v < N; v++) {
            if (h_stash_excess[v] != 0) {
                h_excess[v] += h_stash_excess[v];
                h_stash_excess[v] = 0;
            }
        }

        // Construct active nodes.
        h_active_nodes.clear();
        for (auto u = 0; u < N; u++) {
            if (h_excess[u] > 0 && u != src && u != sink) {
                h_active_nodes.emplace_back(u);
            }
        }
    }

    // Free host memory
    free(h_dist);
    free(h_excess);
    free(h_stash_send);
    free(h_stash_dist);
    free(h_stash_excess);

    // Free device memory
    hipFree(d_dist);
    hipFree(d_excess);
    hipFree(d_stash_dist);
    hipFree(d_stash_send);
    hipFree(d_stash_excess);

    return 0;
}
