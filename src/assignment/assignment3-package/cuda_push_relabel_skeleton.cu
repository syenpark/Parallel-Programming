#include "hip/hip_runtime.h"
/**
 * Name: Seoyoung Park
 * Student id:
 * ITSC email: sparkap@connect.ust.hk
 */

#include <cstring>
#include <cstdint>
#include <cstdlib>

#include <vector>
#include <iostream>

#include "cuda_push_relabel.h"

using namespace std;

void pre_flow(int *dist, int64_t *excess, int *cap, int *flow, int N, int src) {
    dist[src] = N;

    for (auto v = 0; v < N; v++) {
        flow[utils::idx(src, v, N)] = cap[utils::idx(src, v, N)];
        flow[utils::idx(v, src, N)] = -flow[utils::idx(src, v, N)];
        excess[v] = flow[utils::idx(src, v, N)];
    }
}

int push_relabel(int blocks_per_grid, int threads_per_block, int N, int src, int sink, int *cap, int *flow) {
    size_t size_in_int = N * sizeof(int);
    size_t size_n_int64_t = N * sizeof(int64_t);

    // Allocate Vectors in host memory (CPU)
    int *h_dist = (int *) calloc(N, sizeof(int));
    int *h_stash_dist = (int *) calloc(N, sizeof(int));
    auto *h_excess = (int64_t *) calloc(N, sizeof(int64_t));
    auto *h_stash_excess = (int64_t *) calloc(N, sizeof(int64_t));

    //printf("%d %d\n", sizeof(int), sizeof(int64_t));
    /*
    // Allocate Vectors in device memory (GPU)
    int *d_dist = (int *) calloc(N, sizeof(int));
    int *d_stash_dist = (int *) calloc(N, sizeof(int));
    auto *d_excess = (int64_t *) calloc(N, sizeof(int64_t));
    auto *d_stash_excess = (int64_t *) calloc(N, sizeof(int64_t));
    int *d_stash_send = (int *) calloc(N * N, sizeof(int));

    hipMalloc((void**) &d_dist, size);
    hipMalloc((void**) &d_stash_dist, size);
    hipMalloc((void**) &d_excess, size);
    hipMalloc((void**) &d_stash_excess, size);
    hipMalloc((void**) &d_stash_send, size);
     */


    // PreFlow.
    pre_flow(h_dist, h_excess, cap, flow, N, src);

    vector<int> active_nodes;
    int *stash_send = (int *) calloc(N * N, sizeof(int));

    for (auto u = 0; u < N; u++) {
        if (u != src && u != sink) {
            active_nodes.emplace_back(u);
        }
    }

    // Four-Stage Pulses.
    while (!active_nodes.empty()) {
        // Stage 1: push.
        for (auto u : active_nodes) {
            for (auto v = 0; v < N; v++) {
                auto residual_cap = cap[utils::idx(u, v, N)] -
                                    flow[utils::idx(u, v, N)];
                if (residual_cap > 0 && h_dist[u] > h_dist[v] && h_excess[u] > 0) {
                    stash_send[utils::idx(u, v, N)] = std::min<int64_t>(h_excess[u], residual_cap);
                    h_excess[u] -= stash_send[utils::idx(u, v, N)];
                }
            }
        }
        for (auto u : active_nodes) {
            for (auto v = 0; v < N; v++) {
                if (stash_send[utils::idx(u, v, N)] > 0) {
                    flow[utils::idx(u, v, N)] += stash_send[utils::idx(u, v, N)];
                    flow[utils::idx(v, u, N)] -= stash_send[utils::idx(u, v, N)];
                    h_stash_excess[v] += stash_send[utils::idx(u, v, N)];
                    stash_send[utils::idx(u, v, N)] = 0;
                }
            }
        }

        // Stage 2: relabel (update dist to stash_dist).
        memcpy(h_stash_dist, h_dist, N * sizeof(int));
        for (auto u : active_nodes) {
            if (h_excess[u] > 0) {
                int min_dist = INT32_MAX;
                for (auto v = 0; v < N; v++) {
                    auto residual_cap = cap[utils::idx(u, v, N)] - flow[utils::idx(u, v, N)];
                    if (residual_cap > 0) {
                        min_dist = min(min_dist, h_dist[v]);
                        h_stash_dist[u] = min_dist + 1;
                    }
                }
            }
        }

        // Stage 3: update dist.
        swap(h_dist, h_stash_dist);

        // Stage 4: apply excess-flow changes for destination vertices.
        for (auto v = 0; v < N; v++) {
            if (h_stash_excess[v] != 0) {
                h_excess[v] += h_stash_excess[v];
                h_stash_excess[v] = 0;
            }
        }

        // Construct active nodes.
        active_nodes.clear();
        for (auto u = 0; u < N; u++) {
            if (h_excess[u] > 0 && u != src && u != sink) {
                active_nodes.emplace_back(u);
            }
        }
    }

    free(h_dist);
    free(h_stash_dist);
    free(h_excess);
    free(h_stash_excess);
    free(stash_send);

    return 0;
}
