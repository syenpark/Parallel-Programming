#include "hip/hip_runtime.h"
/**
 * Name: Seoyoung Park
 * Student id:
 * ITSC email: sparkap@connect.ust.hk
 */
// #define int int64_t
#include <cstring>
#include <cstdint>
#include <cstdlib>

#include <vector>
#include <iostream>

#include "cuda_push_relabel.h"

using namespace std;

__device__ int idx(int x, int y, int n) {
    return x * n + y;
}

__global__ void pre_flow(int *dist, unsigned long long int *excess, int *cap, int *flow, int N, int src) {
    dist[src] = N;
    int num_threads = blockDim.x * gridDim.x;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    for (auto v = tid; v < N; v += num_threads) {
        flow[idx(src, v, N)] = cap[idx(src, v, N)];
        flow[idx(v, src, N)] = -flow[idx(src, v, N)];
        excess[v] = flow[idx(src, v, N)];
    }
}

__global__ void push(int *dist, unsigned long long int *excess, int *cap, int *flow, int N, int src, int *active_nodes, int count, unsigned long long int *stash_excess) {
    int num_threads = blockDim.x * gridDim.x;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    for (int i = tid; i < count; i += num_threads) {
        int u = active_nodes[i];

        for (auto v = 0; v < N; v++) {
            long long int residual_cap = cap[idx(u, v, N)] - flow[idx(u, v, N)];

            if (residual_cap > 0 && dist[u] > dist[v] && excess[u] > 0) {
                unsigned long long int tmp = min(excess[u], residual_cap);
                excess[u] -= tmp;
                atomicAdd(flow + idx(u, v, N), tmp);
                atomicSub(flow + idx(v, u, N), tmp);
                atomicAdd(stash_excess + v, tmp);
            }
        }
    }
}


__global__ void relabel(int N, int src, int *dist, unsigned long long int *excess, int *cap, int *flow, int *active_nodes, int count, int *stash_dist) {
    int start = blockDim.x * blockIdx.x + threadIdx.x;
    int elementSkip = blockDim.x * gridDim.x;

    for (int i = start; i < count; i += elementSkip) {
        int u = active_nodes[i];

        if (excess[u] > 0) {
            int min_dist = INT32_MAX;

            for (auto v = 0; v < N; v++) {
                auto residual_cap = cap[idx(u, v, N)] - flow[idx(u, v, N)];
                if (residual_cap > 0) {
                    min_dist = atomicMin(dist + v, min_dist);
                    stash_dist[u] = min_dist + 1;
                }
            }
        }
    }
}

__global__ void apply_changes(int N, unsigned long long int *excess, unsigned long long int *stash_excess){
    int start = blockDim.x * blockIdx.x + threadIdx.x;
    int elementSkip = blockDim.x * gridDim.x;
    for (int v = start; v < N; v += elementSkip) {
        if (stash_excess[v] != 0) {
            excess[v] += stash_excess[v];
            stash_excess[v] = 0;
        }
    }
}

int push_relabel(int blocks_per_grid, int threads_per_block, int N, int src, int sink, int *cap, int *flow) {
    int *dist = (int *) calloc(N, sizeof(int));
    int *stash_dist = (int *) calloc(N, sizeof(int));
    unsigned long long int *excess = (unsigned long long int *) calloc(N, sizeof(unsigned long long));
    unsigned long long int *stash_excess = (unsigned long long int *) calloc(N, sizeof(unsigned long long));

    int *dist_d, *stash_dist_d;
    hipMalloc(&dist_d, N * sizeof(int));
    hipMalloc(&stash_dist_d, N * sizeof(int));

    unsigned long long int *excess_d, *stash_excess_d;
    hipMalloc(&excess_d, N * sizeof(unsigned long long));
    hipMalloc(&stash_excess_d, N * sizeof(unsigned long long));

    int *cap_d, *flow_d;
    hipMalloc(&cap_d, N * N * sizeof(int));
    hipMalloc(&flow_d, N * N * sizeof(int));

    hipMemcpy(cap_d, cap, sizeof(int) * N * N, hipMemcpyHostToDevice);
    pre_flow<<<blocks_per_grid, threads_per_block>>>(dist_d, excess_d, cap_d, flow_d, N, src);

    vector<int> active_nodes;
    int *active_nodes_d;
    hipMalloc(&active_nodes_d, N * sizeof(int));

    unsigned long long int *stash_send = (unsigned long long int *) calloc(N * N, sizeof(unsigned long long));
    for (auto u = 0; u < N; u++) {
        if (u != src && u != sink) {
            active_nodes.emplace_back(u);
        }
    }

    while (!active_nodes.empty()) {
        int count = active_nodes.size();

        hipMemcpy(active_nodes_d, active_nodes.data(), sizeof(int) * count, hipMemcpyHostToDevice);
        push<<<blocks_per_grid, threads_per_block>>>(dist_d, excess_d, cap_d, flow_d, N, src, active_nodes_d, count, stash_excess_d);

        hipMemcpy(stash_dist_d, dist_d, N * sizeof(int), hipMemcpyDeviceToDevice);
        relabel<<<blocks_per_grid, threads_per_block>>>(N, src, dist_d, excess_d, cap_d, flow_d, active_nodes_d, count, stash_dist_d);

        // Stage 3. Update
        swap(dist_d, stash_dist_d);

        // Stage 4: apply excess-flow changes for destination vertices.
        apply_changes<<<blocks_per_grid, threads_per_block>>>(N, excess_d, stash_excess_d);
        hipMemcpy(excess, excess_d, sizeof(unsigned long long) * N, hipMemcpyDeviceToHost);
        hipMemcpy(flow, flow_d, sizeof(int) * N * N, hipMemcpyDeviceToHost);

        active_nodes.clear();
        for (auto u = 0; u < N; u++) {
            if (excess[u] > 0 && u != src && u != sink) {
                active_nodes.emplace_back(u);
            }
        }
    }

    free(dist);
    free(stash_dist);
    free(excess);
    free(stash_excess);
    free(stash_send);

    // hipFree(dist_d);
    // hipFree(stash_dist_d);
    // hipFree(excess_d);
    // hipFree(stash_excess_d);
    // hipFree(cap_d);
    // hipFree(flow_d);
    // hipFree(active_nodes_d);

    return 0;
}
