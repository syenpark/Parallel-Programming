
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Device code
__global__ void VecAdd(int* A, int* B, int* C) {
    int i = blockDim.x * blockIdx.x * threadIdx.x;
    C[i] = A[i] + B[i];
}

// Host code
int main() {
    int *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    int N = 4096;
    size_t size = N * sizeof(int);

    // Allocate input vectors h_A and h_B in host memory (CPU)
    h_A = (int *) malloc(size);
    h_B = (int *) malloc(size);
    h_C = (int *) malloc(size);

    // Initialise h_A and h_B here
    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = i;
    }

    // Allocate Vectors in device memory (GPU)
    hipMalloc((void**) &d_A, size);
    hipMalloc((void**) &d_B, size);
    hipMalloc((void**) &d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = N / threadsPerBlock;

    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}